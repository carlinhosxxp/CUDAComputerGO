/* 
 Percurso na GameTree - Jogo GO (Adaptado) - utilizando algoritmo MONTE CARLO TREE SEARCH (MCTS)
 Vers�o Paralela (CUDA)
 Carlos Henrique Rorato Souza - Computa��o Paralela (2022-1)
 Varia��o do jogo: 
     - Tabuleiro de tamanho N*N intersec��es
     - Captura de somente uma pe�a por vez, cercada na horizontal e vertical
     - C�lculo do score baseado na quantidade de pe�as pretas e brancas restantes
 � armazenado somente o estado atual de jogo, e as simula��es s�o feitas a partir deste estado.
 Paraleliza��o: cada bloco far� o processamento (simula��es) referente � um n� filho da raiz.
*/

/* 
 Defini��es iniciais:
	- N � o tamanho do tabuleiro
	- qtd_jogoadas define a quantidade de jogadas (n�veis da �rvore)
	- num_simulacoes define a quantidade de simula��es que o MCTS far� para cada n�
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 19
#define qtd_jogadas N*N
#define num_simulacoes 100000

/*
 Estrutura Estado, que representa um n� da �rvore, com:
	- Tabuleiro
	- Peca (qual pe�a foi colocada nesta jogada)
	- Score (pontua��o, calculada em fun��o espec�fica)
	- Linha, Coluna (em qual posi��o essa pe�a foi colocada)
	- N�vel (o n�vel da �rvore no qual o n� est�)
*/
struct Estado{
	char tabuleiro[N][N];
	char peca;
	int score;
	int linha;
	int coluna;
	int nivel;
};

/*
 Fun��o que faz a gera��o de n�meros pseudoaleat�rios em CUDA,
 para uso na etapa de simula��o.
 Para a inicializa��o, informamos a semente para gera��o dos n�meros.
*/
__device__ void random(int* resultado, int limite) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState_t state;
	hiprand_init(clock(), id, 0, &state);
	*resultado = hiprand(&state) % limite;
}

/*
 Fun��o que inicializa um estado com os valores padr�o, 
 sem pai e sem filhos, com tabuleiro vazio.
 Como ela � executada tanto na GPU quanto na CPU, est�o dispon�veis as duas vers�es.
*/
__device__ void inicializaEstadoPadrao(struct Estado *s){
	int i,j;
	for(i=0; i<N; i++) for(j=0; j<N; j++) s->tabuleiro[i][j] = '-';
	s->score = 0;
	s->peca = '-';
	s->linha = 0;
	s->coluna = 0;
	s->nivel = 0;
}

void inicializaEstadoPadraoHost(struct Estado *s){
	int i,j;
	for(i=0; i<N; i++) for(j=0; j<N; j++) s->tabuleiro[i][j] = '-';
	s->score = 0;
	s->peca = '-';
	s->linha = 0;
	s->coluna = 0;
	s->nivel = 0;
}

/*
 Fun��o que calcula o score, percorrendo o tabuleiro
 do estado. Calcula-se o score do jogador que usa as
 pedras brancas. O c�lculo � feito a partir da diferen�a
 entre as pe�as brancas e pretas restantes no tabuleiro.
 Como ela � executada tanto na GPU quanto na CPU, est�o dispon�veis as duas vers�es.
*/
__device__ void calculaScore(struct Estado *s){
	int i,j,p=0,b=0;
	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			if(s->tabuleiro[i][j] != '-'){
				if(s->tabuleiro[i][j] == 'p') p++;
				else b++;
			}
		
		}
	} 	
	s->score = b - p; 
}

void calculaScoreHost(struct Estado *s){
	int i,j,p=0,b=0;
	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			if(s->tabuleiro[i][j] != '-'){
				if(s->tabuleiro[i][j] == 'p') p++;
				else b++;
			}
		
		}
	} 	
	s->score = b - p; 
}

/*
 Fun��o que insere uma pe�a no tabuleiro e verifica se essa inser��o
 captura alguma pe�a da cor oposta, verificando as quatro dire��es
 (cima, baixo, direita e esqueda). Caso alguma pe�a esteja nas bordas
 do tabuleiro, ela j� � considerada cercada na dire��o da(s) borda(s).
 A fun��o verifica tamb�m se o movimento � suicida, isto �, se a pe�a 
 foi inserida numa posi��o onde � capturada.
 Como ela � executada tanto na GPU quanto na CPU, est�o dispon�veis as duas vers�es.
*/
__device__ void fazMovimento(struct Estado *s, char peca, int i, int j){ //peca pode ser "b" ou "p"
	int contador;
	char oponente = peca == 'b'? 'p': 'b';
	
	if(s->tabuleiro[i][j] == '-'){
		s->tabuleiro[i][j] = peca;
	
		/* olhado para cima (j-1) e verificando se a inser��o da pe�a cercou a pe�a de cima */
		contador = 0;
		if(j-1>=0 && s->tabuleiro[i][j-1] == oponente){
			if (j-1 + 1 < N) {if(s->tabuleiro[i][j-1 + 1] == peca) contador++;} else contador++;
			if (j-1 - 1 >=0) {if(s->tabuleiro[i][j-1 - 1] == peca) contador++;} else contador++;
			if (i+1 < N) {if(s->tabuleiro[i + 1][j-1] == peca) contador++;} else contador++;
			if (i-1 >=0) {if(s->tabuleiro[i - 1][j-1] == peca) contador++;} else contador++;
			
			/* Se a pe�a est� cercada, � capturada */
			if(contador == 4) s->tabuleiro[i][j-1] = '-';
		}
		
		/* olhado para baixo (j+1) e verificando se a inser��o da pe�a cercou a pe�a de baixo */
		contador = 0;
		if(j+1<N && s->tabuleiro[i][j+1] == oponente){
			if (j+1 + 1 < N) {if(s->tabuleiro[i][j+1 + 1] == peca) contador++;} else contador++;
			if (j+1 - 1 >=0) {if(s->tabuleiro[i][j+1 - 1] == peca) contador++;} else contador++;
			if (i+1 < N) {if(s->tabuleiro[i + 1][j+1] == peca) contador++;} else contador++;
			if (i-1 >=0) {if(s->tabuleiro[i - 1][j+1] == peca) contador++;} else contador++;
			
			/* Se a pe�a est� cercada, � capturada */
			if(contador == 4) s->tabuleiro[i][j+1] = '-';
		}
		
		/* olhado para a esquerda (i-1) e verificando se a inser��o da pe�a cercou a pe�a da esquerda */
		contador = 0;
		if(i-1>=0 && s->tabuleiro[i-1][j] == oponente){
			if (j + 1 < N) {if(s->tabuleiro[i - 1][j + 1] == peca) contador++;} else contador++;
			if (j - 1 >=0) {if(s->tabuleiro[i - 1][j - 1] == peca) contador++;} else contador++;
			if (i - 1 + 1 < N) {if(s->tabuleiro[i -1 + 1][j] == peca) contador++;} else contador++;
			if (i - 1 - 1 >=0) {if(s->tabuleiro[i -1 - 1][j] == peca) contador++;} else contador++;
			
			/* Se a pe�a est� cercada, � capturada */
			if(contador == 4) s->tabuleiro[i-1][j] = '-';
		}
		
		/* olhado para a direita (i+1) e verificando se a inser��o da pe�a cercou a pe�a da direita */
		contador = 0;
		if(i+1<N && s->tabuleiro[i+1][j] == oponente){
			if (j + 1 < N) {if(s->tabuleiro[i + 1][j + 1] == peca) contador++;} else contador++;
			if (j - 1 >=0) {if(s->tabuleiro[i + 1][j - 1] == peca) contador++;} else contador++;
			if (i + 1 + 1 < N) {if(s->tabuleiro[i + 1 + 1][j] == peca) contador++;} else contador++;
			if (i + 1 - 1 >=0) {if(s->tabuleiro[i + 1 - 1][j] == peca) contador++;} else contador++;
			
			/* Se a pe�a est� cercada, � capturada */
			if(contador == 4) s->tabuleiro[i+1][j] = '-';
		}
		
		/* Avaliando se a jogada � suicida, ou seja, se a pe�a foi inserida numa posi��o onde � capturada */
		contador = 0;
		if(j + 1 < N){  if(s->tabuleiro[i][j+1] == oponente) contador++; } else contador++;
		if(j - 1 >= 0){ if(s->tabuleiro[i][j-1] == oponente) contador++; } else contador++;
		if(i + 1 < N){  if(s->tabuleiro[i+1][j] == oponente) contador++; } else contador++;
		if(i - 1 >= 0){ if(s->tabuleiro[i-1][j] == oponente) contador++; } else contador++;
		
		/* Se a pe�a est� cercada, � capturada */
		if(contador == 4) s->tabuleiro[i][j] = '-';
		
		/* Ao final, calcula o score */
		calculaScore(s);
	}
}


void fazMovimentoHost(struct Estado *s, char peca, int i, int j){ //peca pode ser "b" ou "p"
	int contador;
	char oponente = peca == 'b'? 'p': 'b';
	
	if(s->tabuleiro[i][j] == '-'){
		s->tabuleiro[i][j] = peca;
	
		/* olhado para cima (j-1) e verificando se a inser��o da pe�a cercou a pe�a de cima */
		contador = 0;
		if(j-1>=0 && s->tabuleiro[i][j-1] == oponente){
			if (j-1 + 1 < N) {if(s->tabuleiro[i][j-1 + 1] == peca) contador++;} else contador++;
			if (j-1 - 1 >=0) {if(s->tabuleiro[i][j-1 - 1] == peca) contador++;} else contador++;
			if (i+1 < N) {if(s->tabuleiro[i + 1][j-1] == peca) contador++;} else contador++;
			if (i-1 >=0) {if(s->tabuleiro[i - 1][j-1] == peca) contador++;} else contador++;
			
			/* Se a pe�a est� cercada, � capturada */
			if(contador == 4) s->tabuleiro[i][j-1] = '-';
		}
		
		/* olhado para baixo (j+1) e verificando se a inser��o da pe�a cercou a pe�a de baixo */
		contador = 0;
		if(j+1<N && s->tabuleiro[i][j+1] == oponente){
			if (j+1 + 1 < N) {if(s->tabuleiro[i][j+1 + 1] == peca) contador++;} else contador++;
			if (j+1 - 1 >=0) {if(s->tabuleiro[i][j+1 - 1] == peca) contador++;} else contador++;
			if (i+1 < N) {if(s->tabuleiro[i + 1][j+1] == peca) contador++;} else contador++;
			if (i-1 >=0) {if(s->tabuleiro[i - 1][j+1] == peca) contador++;} else contador++;
			
			/* Se a pe�a est� cercada, � capturada */
			if(contador == 4) s->tabuleiro[i][j+1] = '-';
		}
		
		/* olhado para a esquerda (i-1) e verificando se a inser��o da pe�a cercou a pe�a da esquerda */
		contador = 0;
		if(i-1>=0 && s->tabuleiro[i-1][j] == oponente){
			if (j + 1 < N) {if(s->tabuleiro[i - 1][j + 1] == peca) contador++;} else contador++;
			if (j - 1 >=0) {if(s->tabuleiro[i - 1][j - 1] == peca) contador++;} else contador++;
			if (i - 1 + 1 < N) {if(s->tabuleiro[i -1 + 1][j] == peca) contador++;} else contador++;
			if (i - 1 - 1 >=0) {if(s->tabuleiro[i -1 - 1][j] == peca) contador++;} else contador++;
			
			/* Se a pe�a est� cercada, � capturada */
			if(contador == 4) s->tabuleiro[i-1][j] = '-';
		}
		
		/* olhado para a direita (i+1) e verificando se a inser��o da pe�a cercou a pe�a da direita */
		contador = 0;
		if(i+1<N && s->tabuleiro[i+1][j] == oponente){
			if (j + 1 < N) {if(s->tabuleiro[i + 1][j + 1] == peca) contador++;} else contador++;
			if (j - 1 >=0) {if(s->tabuleiro[i + 1][j - 1] == peca) contador++;} else contador++;
			if (i + 1 + 1 < N) {if(s->tabuleiro[i + 1 + 1][j] == peca) contador++;} else contador++;
			if (i + 1 - 1 >=0) {if(s->tabuleiro[i + 1 - 1][j] == peca) contador++;} else contador++;
			
			/* Se a pe�a est� cercada, � capturada */
			if(contador == 4) s->tabuleiro[i+1][j] = '-';
		}
		
		/* Avaliando se a jogada � suicida, ou seja, se a pe�a foi inserida numa posi��o onde � capturada */
		contador = 0;
		if(j + 1 < N){  if(s->tabuleiro[i][j+1] == oponente) contador++; } else contador++;
		if(j - 1 >= 0){ if(s->tabuleiro[i][j-1] == oponente) contador++; } else contador++;
		if(i + 1 < N){  if(s->tabuleiro[i+1][j] == oponente) contador++; } else contador++;
		if(i - 1 >= 0){ if(s->tabuleiro[i-1][j] == oponente) contador++; } else contador++;
		
		/* Se a pe�a est� cercada, � capturada */
		if(contador == 4) s->tabuleiro[i][j] = '-';
		
		/* Ao final, calcula o score */
		calculaScoreHost(s);
	}
}

/*
 Fun��o auxiliar que copia o tabuleiro de um n� para outro.
*/
__device__ void copiarTabuleiro(struct Estado original, struct Estado *copia){
	int i,j;
	for(i=0; i<N; i++){
		for(j=0; j<N; j++){
			copia->tabuleiro[i][j] = original.tabuleiro[i][j];
		}
	}
}

/*
 Fun��o auxiliar para o MCTS que simula o caminho at� o final da �rvore (fim de jogo),
 sorteando caminhos na �rovore a partir do estado s (jogadas aleat�rias)
 e retornando ao final o score desse jogo.
*/
__device__ void simulacaoMCTS(struct Estado s, int *resultado){
	char peca = 'p';
	int i,j;
	
	for(i = 0; i < N*N - s.nivel; i++){		
		/* Sorteia-se uma posi��o v�lida no tabuleiro e se faz a jogada */
		do{
			random(&j,N*N);
		}while(s.tabuleiro[j/N][j%N] != '-');
		fazMovimento(&s, peca,j/N,j%N);
		
		/* Inverte-se a pe�a a ser jogada no pr�ximo n�vel */
		peca = peca == 'b'? 'p': 'b';
	}
	calculaScore(&s);
	*resultado = s.score;
}

/*
 Fun��o MCTS (Monte Carlo Tree Search), que recebe o estado atual da �rvore e
 preenche o vetor de scores, com o somat�rio de scores da simula��o (para cada n�).
 Este algoritmo constr�i somente uma �rvore parcial, com um n� raiz e seus filhos.
 Est� baseado na seguinte sequ�ncia: sele��o da raiz e expans�o dos filhos do n� raiz,
 simula��o de caminhos de jogo aleat�rios para cada n� que foi gerado na expans�o,
 propaga��o do score desta configura��o de jogo e a sele��o da jogada (caminho
 na �rvore) que trar� o score mais favor�vel a partir das simula��es. Estas duas �ltimas etapas
 n�o s�o feitas nessa fun��o, mas dentro da pr�pria fun��o principal, a partir do vetor de scores.
 Ela � executada na GPU. Nesse contexto, a solu��o foi estruturada de maneira que cada bloco
 processe as opera��es referentes a um n� filho.
 Os filhos n�o s�o armazenados, mas a simula��o indicar� a melhor jogada, que ser� concretizada
 dentro do tabuleiro do jogo (estado atual), na fun��o principal.
*/
__global__ void mcts(struct Estado gameTree, int *scores){
	
	/* ETAPA 1 - SELE��O: seleciona o n� raiz e declara/inicializa vari�veis importantes para a fun��o */
	int i = 0;
	int k = 0;
	int stride = blockDim.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	struct Estado temp;

	/* ETAPA 2 - EXPANS�O: faz a expans�o de todos os filhos do n� selecionado (raiz) - esta � a pol�tica de expans�o */
	/* Cada bloco ser� um filho do n� raiz, armazenado de forma tempor�ria */
	inicializaEstadoPadrao(&temp);
	
	/* O filho continua o jogo do pai */
	copiarTabuleiro(gameTree,&temp);
	
	if(temp.tabuleiro[blockIdx.x/N][blockIdx.x%N] == '-'){
		fazMovimento(&temp,'b',blockIdx.x/N,blockIdx.x%N);

		/* ETAPA 3 - SIMULA��O: faz-se a soma de diversas simula��es para cada n� que foi expandido */
		/* Faz em passadas, de forma que em um bloco sejam feitas todas as simula��es do n� */

		scores[idx] = 0;

		for(i = threadIdx.x; i < num_simulacoes; i+= stride){
			k = 0;
			simulacaoMCTS(temp,&k);
			scores[idx] += k;
		}
		
	}else{
		/* Se n�o for poss�vel fazer o movimento e as simula��es, marca-se essa posi��o com -666, fazendo com que a m�dia desse n� n�o seja escolhida. */
		scores[idx]  = -666;
	}
}

int main(){
	struct Estado s;
	int *scores;
	
	int l,m,k;
	int c1,c2;
	int soma;
	int jogadas;
	float maior;
	float media;

	clock_t tempo;
	size_t threadsPerBlock = 256;
	size_t numberOfBlocks = N*N;
	hipError_t err;

	int tam_scores = numberOfBlocks * threadsPerBlock;

	printf("GameTree - Jogo GO (Adaptado) - Percurso com MCTS (Monte Carlo Tree Search)\n");
	printf("Tabuleiro %d x %d.\n",N,N);
	printf("Considerando %d jogadas.\n",qtd_jogadas);
	printf("Serao feitas %d simulacoes para cada no expandido.\n\n",num_simulacoes);
	
	/* Trabalhando com a Mem�ria Unificada - cada thread ter� uma posi��o do vetor de scores */
	err = hipMallocManaged(&scores, tam_scores * sizeof(int));
	if(err != hipSuccess) printf("Error: %s\n",hipGetErrorString(err));

	printf("INICIANDO O JOGO:\n");
	
	/* Inicializando e imprimindo o primeiro estado de jogo - raiz da �rvore */
	inicializaEstadoPadraoHost(&s);
	jogadas = 0;
	
	for(l = 0; l < N; l++){
		for (m = 0; m < N; m++) printf("%c ", s.tabuleiro[l][m]);
		printf("\n");
	}
	
	while(jogadas + 1 <= qtd_jogadas){

		/* Inicializando/resetando o vetor de scores */
		for(l = 0; l < tam_scores; l++) scores[l] = -999;
		
		/* Coleta a jogada, faz o movimento e imprime o tabuleiro */	
		printf("Jogada (p) - linha e coluna: ");
		scanf("%d%d",&c1,&c2);
		
		fazMovimentoHost(&s,'p',c1,c2);
		s.nivel++;
		
		for(l = 0; l < N; l++){
			for (m = 0; m < N; m++) printf("%c ", s.tabuleiro[l][m]);
			printf("\n");
		}
		
		jogadas++;
		
		/* Fazendo o MCTS com a raiz da �rvore */
		if(jogadas + 1 <= qtd_jogadas){
			printf("Fazendo MCTS...\n");

			tempo = clock();

			/* Definindo o numero de blocos/threads e fazendo a chamada do kernel */
			mcts<<<numberOfBlocks,threadsPerBlock>>>(s,scores);
			hipDeviceSynchronize();

			/* Verifica��o de erros na chamada do kernel */
			err = hipGetLastError();
			if(err != hipSuccess) printf("Error: %s\n",hipGetErrorString(err));

			/* ETAPAS FINAIS DO MCTS - PROPAGA��O: o valor do score das simula��es � utilizado para definir o melhor n� */
			/* Obs: dada a forma como o algoritmo foi implementado, n�o � necess�ria uma propaga��o recursiva, com percurso na �rvore */
			/* Ao fim, descobre o melhor filho do n� raiz (cujo �ndice ser� armazenado em k) */
			maior = -999.0;
			k = 0;

			for(l = 0; l < tam_scores; l+=256){
				soma = 0;

				/* != -999 - quer dizer que a thread fez a simula��o */
				for(m = l; m < l+256; m++) if(scores[m] != -999) soma+=scores[m];

				media = (float)soma / num_simulacoes; 

				printf("Media do bloco %d: %f\n",l/256,media);
				if(media > maior){
					maior = media;
					k = l/256;
				}
			} 

			tempo = clock() - tempo;


			/* Imprimindo o resultado do MCTS */
			printf("Resultado do MCTS: indice %d. Tempo: %lf s.\n",k,((double)tempo)/((CLOCKS_PER_SEC)));
			printf("Jogada (b):\n");

			/* Efetivando o melhor movimento encontado na busca e imprimindo o tabuleiro */
			fazMovimentoHost(&s,'b',k/N,k%N);
			s.peca = 'b';
			s.linha = k/N;
			s.coluna = k%N;
			s.nivel++;

			for(l = 0; l < N; l++){
				for (m = 0; m < N; m++) printf("%c ", s.tabuleiro[l][m]);
				printf("\n");
			}

			jogadas++;
		}	

	}
	
	printf("Jogo finalizado - score da maquina (b): %d\n",s.score);

	hipFree(scores);
	return 0;
}
